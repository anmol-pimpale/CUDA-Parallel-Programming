
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2000

__global__ void add_matrix(int* d_C, int* d_A, int* d_B, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < size && j < size) {
        d_C[i * size + j] = d_A[i * size + j] + d_B[i * size + j];
    }
}

int main() {
    // Allocate memory for matrices A, B, C on host
    int** A = (int**)malloc(N * sizeof(int*));
    int** B = (int**)malloc(N * sizeof(int*));
    int** C = (int**)malloc(N * sizeof(int*));

    for (int i = 0; i < N; i++) {
        A[i] = (int*)malloc(N * sizeof(int));
        B[i] = (int*)malloc(N * sizeof(int));
        C[i] = (int*)malloc(N * sizeof(int));
    }

    // Initialize matrices A and B with random values
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = rand() % 10;
            B[i][j] = rand() % 10;
        }
    }

    // Allocate memory for matrices A, B, C on device
    int* d_A;
    int* d_B;
    int* d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(int)); //why we use N * N because integer size is 4 byte thats why provide N 8 N
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy data from host to device
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hipMemcpy(&d_A[i * N + j], &A[i][j], sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(&d_B[i * N + j], &B[i][j], sizeof(int), hipMemcpyHostToDevice);
        }
    }

    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing GPU execution
    hipEventRecord(start);

    // Launch the kernel with the calculated number of blocks and threads per block
    int blockSize = 16;
    int numBlocks = (N + blockSize - 1) / blockSize;
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 numBlocks2D(numBlocks, numBlocks);
    add_matrix<<<numBlocks2D, threadsPerBlock>>>(d_C, d_A, d_B, N);
    hipDeviceSynchronize();

    // Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, stop);

    // Copy the result back from the device
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hipMemcpy(&C[i][j], &d_C[i * N + j], sizeof(int), hipMemcpyDeviceToHost);
        }
    }

    // // Print the result
    // for (int i = 0; i < N; i++) {
    //     for (int j = 0; j < N; j++) {
    //         printf("%d ", C[i][j]);
    //     }
    //     printf("\n");
    // }

    // Print time taken by GPU
    printf("Time taken by GPU: %f miliseconds\n", miliseconds);

    // Free memory
    for (int i = 0; i < N; i++) {
        free(A[i]);
        free(B[i]);
        free(C[i]);
    }
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}