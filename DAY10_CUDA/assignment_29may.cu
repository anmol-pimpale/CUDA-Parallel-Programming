#include <stdio.h>
#include <hip/hip_runtime.h>
#include<time.h>

__global__ void kernel2(float *A, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    A[idx] = A[idx] * A[idx];
  }
}

__global__ void kernel1(float *A, float *B, int n) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < n) {
    A[idx] = A[idx] + B[idx];
  }
  __syncthreads();
  kernel2<<<1,1>>>(A, n);
  // __syncthreads();
}

int main() {
  int n = 100000;
  float* A = (float*)malloc(n * sizeof(float));
  float* B = (float*)malloc(n * sizeof(float));

  float* dev_a;
  float* dev_b;

  hipMalloc((void **)&dev_a, n * sizeof(float));
  hipMalloc((void **)&dev_b, n * sizeof(float));

  // Initialize arrays A and B on the host (CPU)
  for (int i = 0; i < n; ++i) {
    A[i] = i * 0.1f;
    B[i] = i * 0.1f;
  }

  // Start timing
  clock_t start_time = clock();


  // Copy data from host to device
  hipMemcpy(dev_a, A, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, B, n * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel 1: calculate element-wise sum (A+B) on device
  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  kernel1<<<numBlocks, blockSize>>>(dev_a, dev_b, n);

  // Copy final result in A back to host
  hipMemcpy(A, dev_a, n * sizeof(float), hipMemcpyDeviceToHost);

   // Stop timing
   clock_t end_time = clock();
   double execution_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;
 

  // Launch kernel 2: square each element of A on device
  // kernel2<<<numBlocks, blockSize>>>(dev_a, n);

  // // Copy final result in A back to host
  // cudaMemcpy(A, A, n * sizeof(float), cudaMemcpyDeviceToHost);

  // // Print final results (now accessible on the host)
  // printf("Square elements of (A+B):\n");
  // for (int i = 0; i < n; ++i) {
  //   printf("%f ", A[i]);
  // }
  // printf("\n");
  
  // Print execution time
  printf("Execution time: %f seconds\n", execution_time);


  free(A);
  free(B);
  hipFree(dev_a);
  hipFree(dev_b);

  return 0;
}