#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM 90000

// CUDA kernel function for printing prime numbers
__global__ void printPrimeNumbers(int start, int end) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID < end - start + 1) {
        int num = start + threadID;
        bool isPrime = true;

        for (int i = 2; i * i <= num; i++) {
            if (num % i == 0) {
                isPrime = false;
                break;
            }
        }

        if (isPrime && num > 1) {
            // printf("%d ", num);
        }
    }
}

int main() {
    int host_fromNum = 2;
    int host_toNum = NUM;

    // Declare device variables
    int *device_fromNum;
    int *device_toNum;

    // Allocate memory on the device
    hipMalloc((void**)&device_fromNum, sizeof(int));
    hipMalloc((void**)&device_toNum, sizeof(int));

    // Copy data from host to device
    hipMemcpy(device_fromNum, &host_fromNum, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_toNum, &host_toNum, sizeof(int), hipMemcpyHostToDevice);

    // Calculate the number of threads per block and blocks per grid
    int threadPerBlock = 256;
    int blockPerGrid = (host_toNum - host_fromNum + threadPerBlock - 1) / threadPerBlock;

    // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the CUDA kernel
    printPrimeNumbers<<<blockPerGrid, threadPerBlock>>>(host_fromNum, host_toNum);
    hipDeviceSynchronize();

    // Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print time taken by GPU
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);

    // Free allocated memory
    hipFree(device_fromNum);
    hipFree(device_toNum);

    return 0;
}