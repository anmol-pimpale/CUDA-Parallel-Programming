#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
int main(){
    int deviceCount;
    hipGetDeviceCount( &deviceCount);
    if(deviceCount==0){
        printf("NO CUDA DEVICE FOUND.\n");
        return 1;

    }
    for(int device=0;device<deviceCount;++device){
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties( &deviceProp,device );

        printf("device %d :%s\n",device,deviceProp.name);
        printf("compute compatibility:%d.%d\n",deviceProp.major,deviceProp.minor);
        printf("total global meory:%lu bytes\n",(unsigned long)deviceProp.sharedMemPerBlock);
        printf("warp size:%d\n",deviceProp.warpSize);
        printf("max threads per block :%d\n",deviceProp.maxThreadsPerBlock);
        printf("max threads dimenstion:(%d %d %d)\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
        printf("max grid size:(%d %d %d)\n",deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
        printf("clock rate:%d kHz\n",deviceProp.clockRate);
        printf("memory clock rate:%d kHz\n",deviceProp.memoryClockRate);
        printf("memory bus width:%d bits\n",deviceProp.memoryBusWidth);
        printf("L2 cache size:%d bytes\n",deviceProp.l2CacheSize);
        printf("constant memory size:%lu bytes\n",(unsigned long)deviceProp.totalConstMem);
        printf("texture aligment:%lu bytes\n",(unsigned long)deviceProp.textureAlignment);
        printf("\n");

    }
    return 0;
}
