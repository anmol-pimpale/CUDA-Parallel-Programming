#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int square(int a) {
    return a * a;
}

__global__ void doubleValues(int* data, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int value = data[i];
        int squared_value = square(value);
        data[i] = squared_value * 2;
    }
}

int main() {
    int size = 10;
    int* data_host = new int[size];
    int* data_device;

    for (int i = 0; i < size; ++i) {
        data_host[i] = i;
    }

    hipMalloc(&data_device, size * sizeof(int));
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    doubleValues << <(size + threadPerBlock - 1) / threadPerBlock, threadPerBlock >> >(data_device, size);

    hipDeviceSynchronize();
    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        printf("data[%d]=%d\n", i, data_host[i]);
    }

    hipFree(data_device);
    delete[] data_host;

    return 0;
}