#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>


    __global__ void grandChildKernel(){
        printf("from CDAC\n");
    }

    __global__ void childKernel(){
            grandChildKernel<<<1,1>>>();

            hipDeviceSynchronize();//wait for grandchild to compplete

            printf("hello\n");
        }

    __global__ void parentKernel(){
        childKernel<<<1,1>>>();

        //hipDeviceSynchronize();//wait for child to compplete

        printf("world\n");
    }

    int main(){
        parentKernel<<<1,1>>>();
        hipDeviceSynchronize();//wait for parent to complete
        return 0;
    }