#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

// Global function for CUDA
__global__ void vectorAddition_kernel(int* d_a, int* d_b, int* d_c, int N) {
    int tid = threadIdx.x;
    int i;
    for(i=tid;i<N;i+=blockDim.x){
          d_c[i]=d_a[i]+d_b[i];
}

}

int main() {

    int N = 8;
    int* h_a, * h_b, * h_c;  // Host Variables
    int* d_a, * d_b, * d_c;  // Device Variables

    // Allocate memory for host variables
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Allocate memory for device variables
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Initialize host variables
    for (int i = 0; i < N; i++) {
        h_a[i] = 2;
        h_b[i] = 2;
        h_c[i] = 0;
    }

    // Copy host variables to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    

    //tart timing GPU execution
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);


    // Launch the kernel function
    int blocksize = 1;
    int numThreads = 4; // Calculate the number of blocks
    vectorAddition_kernel<<<blocksize, numThreads>>>(d_a, d_b, d_c, N);



    // stop timing gpu execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float miliseconds=0;
    hipEventElapsedTime(&miliseconds,start,stop);


    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    //pritn time taken by GPU 
    printf("time taken by Gpu :%f miliseconds\n",miliseconds);



    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}