#include<iostream>
#include<hip/hip_runtime.h>

int main() {
    int maxThreadsPerBlock, maxBlocks;

    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    hipDeviceGetAttribute(&maxBlocks, hipDeviceAttributeMaxGridDimX, 0);

    std::cout << "Maximum Threads Per Block: " << maxThreadsPerBlock << std::endl;
    std::cout << "Maximum Blocks: " << maxBlocks << std::endl;

    return 0;
}