#include<stdio.h>

#include<hip/hip_runtime.h>
#define SIZE 90000

__global__ void add_array(int *c,const int*a,const int *b,int size){
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    if (i<size){
        c[i]=a[i]+b[i];
    }
}

int main(){
    const int size = SIZE;
    int a[size];
    int b[size];
    int *d_c;

    // Initialize arrays a and b with some values
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory on the device or GPU
    hipMalloc((void**)&d_c,size * sizeof(int));

    // Copy array a and b to the device
    int *d_a,*d_b;
    hipMalloc((void**)&d_a,size * sizeof(int));
    hipMalloc((void**)&d_b,size * sizeof(int));

    hipMemcpy(d_a ,a ,size * sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_b ,b ,size * sizeof(int) , hipMemcpyHostToDevice);

    // Calculate the number of blocks and threads per block
    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;


    //tart timing GPU execution
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the kernel with the calculated number of blocks and threads per block
    add_array<<<blocks, threadsPerBlock>>>(d_c,d_a,d_b,size);
    hipDeviceSynchronize();
  

    // stop timing gpu execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float miliseconds=0;
    hipEventElapsedTime(&miliseconds,start,stop);


    // Copy the result back from the device
    int *c= (int*)malloc(size * sizeof(int));
    hipMemcpy(c ,d_c ,size * sizeof(int) , hipMemcpyDeviceToHost);

    // Print the result
    for(int i=0;i<size;i++){
        printf("%d :",c[i]);
    }
    printf("\n");

    //pritn time taken by GPU 
    printf("time taken by Gpu :%f miliseconds\n",miliseconds);

    // Calculate the sum of all elements in array c
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += c[i];
    }
    printf("Sum of all elements: %d\n", sum);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(c);

    return 0;
}