#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

// Global function for CUDA
__global__ void vectorAddition_kernel(int* d_a, int* d_b, int* d_c, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        d_c[tid] = d_a[tid] + d_b[tid];
    }
}

int main() {

    int N = 8;
    int* h_a, * h_b, * h_c;  // Host Variables
    int* d_a, * d_b, * d_c;  // Device Variables

    // Allocate memory for host variables
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Allocate memory for device variables
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Initialize host variables
    for (int i = 0; i < N; i++) {
        h_a[i] = 2;
        h_b[i] = 2;
        h_c[i] = 0;
    }

    // Copy host variables to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel function
    int blocksize = 1;
    int numThreads = 4; // Calculate the number of blocks
    vectorAddition_kernel<<<blocksize, numThreads>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    printf("Result: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    // Free device and host memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}