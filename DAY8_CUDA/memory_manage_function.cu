//using manged memory function both CPU and GPU access the managed memory

#include<hip/hip_runtime.h>

#include<stdio.h>

__global__ void printValue(int* data){
    int tid=threadIdx.x + blockIdx.x * blockDim.x;


    //access managed memory directlty from GPU
    printf("Gpu thread %d:value=%d\n",tid,data[tid]);

}

int main(){

    const int N=10;

    //allocate manages memory
    int* data;
    hipMallocManaged(&data ,N * sizeof(int) );

    //initialise data on CPU
    for(int i=0;i<N;++i){
        data[i]= i *2;
    }
    
   // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //launch GPU kernel  to print value
    printValue<<<1,N>>>(data);
    hipDeviceSynchronize();

    // Stop timing GPU execution
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);


    //access managed memory directly from cpu
    for(int i=0;i<N;++i){
        printf("CPU : value =%d\n",data[i]);

    }

    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);


    //free  managed memoroy
    hipFree(data );


    return 0;
}