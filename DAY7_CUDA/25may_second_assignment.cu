#include "hip/hip_runtime.h"
//second assignmet with two global function
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void squareValues(int *a) {
int num=*a;
*a=num*num;

}

__global__ void doubleValues(int* data, int size) {
    // int threadPerBlock = 256;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
   
    if (i < size) {

    int *num=data+i;
    squareValues<<< 1,1 >>>(num);
    hipDeviceSynchronize();

    int doubleSquareValue=(*num)*2;
    printf(" %d * %d * 2 = %d\n",i,i,doubleSquareValue);
    }
}

int main() {
    int size = 10;
    int* data_host = new int[size];
    int* data_device;

    for (int i = 0; i < size; ++i) {
        data_host[i] = i;
    }

    hipMalloc(&data_device, size * sizeof(int));
    hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);
    
    // Start timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
                 
    int threadPerBlock = 256;
    doubleValues << <(size + threadPerBlock - 1) / threadPerBlock, threadPerBlock >> >(data_device, size);

    hipDeviceSynchronize();
    
     // Stop timing GPU execution
     hipEventRecord(stop);
     hipEventSynchronize(stop);
     float milliseconds = 0;
     hipEventElapsedTime(&milliseconds, start, stop);
 
    hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < size; ++i) {
    //     // printf("data[%d]=%d\n", i, data_host[i]);
    // }
    printf("\nTime taken by GPU : %f milliseconds\n", milliseconds);
  
    hipFree(data_device);
    delete[] data_host;

    return 0;
}

//OUTPUT: in above code with two global function code is execute but square of the value is not happend only double the value.