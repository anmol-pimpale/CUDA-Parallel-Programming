#include<stdio.h>

#include<hip/hip_runtime.h>

__global__ void printThreadInfo(){
    int threadID=blockIdx.x * blockDim.x + threadIdx.x;
    printf("ThraedIdx:%d,blockIdx:%d,blockDim.x:%d,Efective Thread ID:%d\n",threadIdx.x,blockIdx.x,blockDim.x,threadID);

}
int main(){
      int numBlocks=5;
      int threadsPerBlock=3;


      printThreadInfo<<<numBlocks,threadsPerBlock>>>();
      hipDeviceSynchronize();
      return 0;

}