#include<stdio.h>

#include<hip/hip_runtime.h>

int main()
{
    const int arraySize = 5;

    //host(cpu) data
    float hostArray[arraySize] = {1.0, 2.0, 3.0, 4.0, 5.0};
    float resultArray[arraySize];

    // Device (gpu) data
    float* deviceArray;
    hipMalloc((void**)&deviceArray, arraySize * sizeof(float));

    //copy data grom cpu to gpu 
    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(float) , hipMemcpyHostToDevice);

    //copy data back from gpu to cpu for result display
    hipMemcpy(resultArray ,deviceArray ,arraySize * sizeof(float) , hipMemcpyDeviceToHost);

    //display result using printf
    printf(" Host Array : ");
    for (int i =0 ; i< arraySize ; ++i)
    {
        printf("%f", hostArray[i]);
    }
    printf("\n");
printf("Original Array array:");
    for (int i =0 ; i< arraySize ; ++i)
    {
        printf("%f", resultArray[i]);
    }
    
    printf("\n");

    // free allocate memory on gpu
    hipFree(deviceArray);
    return 0;

}