#include<stdio.h>

#include<hip/hip_runtime.h>

__global__ void addInteger(int* a,int* b,int* result){
    *result = *a + *b;
    printf("In GPU ......sum is %d\n",*result);
}


int main(){
//Hoat variable
  int host_a=5;
  int host_b=7;
  int host_result=0;


  //device variable

  int* device_a,* device_b, * device_result;

  //allocate memory ont the device

  hipMalloc((void**)&device_a,sizeof(int));
  hipMalloc((void**)&device_b,sizeof(int));
  hipMalloc((void**)&device_result,sizeof(int));

  //copy data from host to device
  hipMemcpy(device_a,  &host_a ,sizeof(int) , hipMemcpyHostToDevice);
  hipMemcpy(device_b,  &host_b ,sizeof(int) , hipMemcpyHostToDevice);

  //lauch the kernel with one block and one thraed
  addInteger <<<1,1024>>> (device_a,device_b,device_result);

  //copy the result from device to host
  hipMemcpy(&host_result ,device_result ,sizeof(int) , hipMemcpyDeviceToHost);

  //display the result
  printf("sum of %d and %d is %d\n",host_a,host_b,host_result);

  //free allocated memory
  hipFree( device_a);
  hipFree( device_b);
  hipFree(device_result );

  return 0;
}


