#include<stdio.h>

#include<hip/hip_runtime.h>


__global__ void add_array(int *c,const int*a,const int *b,int size){
    int i=blockIdx.x * blockDim.x + threadIdx.x;
    if (i<size){
        c[i]=a[i]+b[i];

    }

}


int main(){
    const int size=5;
    int a[size]={1,2,3,4,5};
    int b[size]={1,2,3,4,5};
    int *d_c;


    //allocate memory on the device or gpu
    hipMalloc((void**)&d_c,size * sizeof(int));


    //copy array a and b to the device
    int *d_a,*d_b;
    hipMalloc((void**)&d_a,size * sizeof(int));
    hipMalloc((void**)&d_b,size * sizeof(int));

    hipMemcpy(d_a ,a ,size * sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_b ,b ,size * sizeof(int) , hipMemcpyHostToDevice);

    add_array<<<2,8>>>(d_c,d_a,d_b,size);
    hipDeviceSynchronize();

    //copy the result back from the device
    int *c= (int*)malloc(5 * sizeof(int));
    hipMemcpy(c ,d_c ,size * sizeof(int) , hipMemcpyDeviceToHost);

    //pritn the result

    for(int i=0;i<size;i++){
        printf("%d :",c[i]);
    }
    printf("\n");

    //frree memory

    hipFree(c);
    hipFree(d_a );
    hipFree(d_b );
    hipFree(d_c );




    
  return 0;
}
