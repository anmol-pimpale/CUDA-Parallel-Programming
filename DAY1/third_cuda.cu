#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void kernel(void){
  printf("Hello from GPU\n");
}


void cpu_print(void){
    printf("Hello from CPU\n");
}
int main(){
    kernel<<<1,10>>>();
    

   hipDeviceSynchronize();
    cpu_print();
    
    
    return 0;
}