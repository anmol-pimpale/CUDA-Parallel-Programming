#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void kernel(void){
  printf("Hello from GPU\n");
}


void cpu_print(void){
    printf("Hello from CPU\n");
}
int main(){
    kernel<<<1,1>>>();
    kernel<<<1,1>>>();
    kernel<<<1,1>>>();

   hipDeviceSynchronize();
    cpu_print();
    cpu_print();
    cpu_print();
    
    return 0;
}